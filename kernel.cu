#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <algorithm>
#include "Bosel.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define SIZE 10

#define db(a) cout << #a << " = " << a << endl
#define db2(a, b) cout << #a << " = " << a << " " << #b << " = " << b << endl

void performCPU(string filename)
{
	std::clock_t start = clock();
	double duration;

	ImgFloat imagen(filename.c_str());
	//ImgFloat imagen("lena30.jpg");
	// depth, numColors, initialize
	ImgFloat xGradient(imagen.width(), imagen.height(), 1, 1, 0);
	ImgFloat yGradient(imagen.width(), imagen.height(), 1, 1, 0);
	ImgFloat gradientA(imagen.width(), imagen.height(), 1, 1, 0);
	ImgFloat gradientB(imagen.width(), imagen.height(), 1, 1, 0);

	//imagen.blur(1.5);

	ImgFloat R = imagen.get_channel(0);

	Bosel b;
	b.convolution(R, b.Gx, xGradient);
	b.convolution(R, b.Gy, yGradient);

	b.mergeA(gradientA, xGradient, yGradient);
	b.mergeB(gradientB, xGradient, yGradient);
	
	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	printf("CPU for image %s takes %.2f seconds\n",filename.c_str(), duration);

	(gradientA, gradientB).display("comparaci�n suma ABSs y SQRT");
	//(xGradient, yGradient, gradient).display("Detecci�n de Bordes");
	cout << duration << endl;
	//gradient.display();
}

__device__ void convolution(int coordinate, float* d_arr, float* gradient, int width, int len,int* mask, int* dir, int* pos)
{
	float c = 0;
	for (int ii = 0; ii < 3; ii++)
	{
		for (int jj = 0; jj < 3; jj++)
		{
			int x = coordinate + width * dir[ii * 3 + jj] + pos[jj];
			if (x >= 0 && x < len)
				c += d_arr[x] * mask[ii * 3 + jj];
		}
	}
	gradient[coordinate] = c;
}

__global__ void deviceComputeGradient(float* d_arr, float* gradient, int width, int len,int* mask, int* dir,int* pos) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x * width + y < len)
		convolution(x * width + y, d_arr, gradient, width, len, mask, dir, pos);
}

__global__ void deviceMerge(float* xGradient, float* yGradient, float* target, int width, int len) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int index = x * width + y;
	if (index < len)
		target[index] = abs(xGradient[index]) + abs(yGradient[index]);
}

void performGPU(string filename) 
{
	ImgFloat imagen(filename.c_str());
	
	std::clock_t start = clock();
	double duration;

	
	ImgFloat img_xGradient(imagen.width(), imagen.height(), 1, 1, 0);
	ImgFloat img_yGradient(imagen.width(), imagen.height(), 1, 1, 0);
	ImgFloat result(imagen.width(), imagen.height(), 1, 1, 0);
	
	// depth, numColors, initialize
	int WIDTH = imagen.width();
	int HEIGHT = imagen.height();
	float *arr, *xGradient, *yGradient, *gradient;
	float *d_arr, *d_xGradient, *d_yGradient, *d_gradient;

	arr = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
	xGradient = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
	yGradient = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
	gradient = (float*)malloc(WIDTH * HEIGHT * sizeof(float));

	hipMalloc((void**)&d_arr, WIDTH * HEIGHT * sizeof(float));
	hipMalloc((void**)&d_xGradient, WIDTH * HEIGHT * sizeof(float));
	hipMalloc((void**)&d_yGradient, WIDTH * HEIGHT * sizeof(float));
	hipMalloc((void**)&d_gradient, WIDTH * HEIGHT * sizeof(float));

	for (int i = 0; i < WIDTH; i++)
		for (int j = 0; j < HEIGHT; j++) {
			arr[i * WIDTH + j] = imagen(i, j);
			xGradient[i * WIDTH + j] = 0;
			yGradient[i * WIDTH + j] = 0;
			gradient[i * WIDTH + j] = 0;
		}
 
	hipMemcpy(d_arr, arr, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_xGradient, xGradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_yGradient, yGradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_gradient, gradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
	
	dim3 BLOCKS = dim3(1024, 1024);
	dim3 THREADS = dim3(4, 4);

	/*dim3 BLOCKS(2, 2);
	dim3 THREADS(2, 2);*/
	
	int pos[3] = { -1, 0, 1 };
	int dir[9] = { -1, -1, -1, 0, 0, 0, 1, 1, 1 };
	int Gx[9] = {
		-1, 0, 1,
		-2, 0, 2,
		-1, 0, 1
	};
	int Gy[9] = {
		 1, 2, 1,
		0, 0, 0,
		-1, -2, -1
	};
	int* d_pos, *d_dir, *d_Gx, *d_Gy;
	
	hipMalloc((void**)&d_pos, 3 * sizeof(int));
	hipMalloc((void**)&d_dir, 9 * sizeof(int));
	hipMalloc((void**)&d_Gx, 9 * sizeof(int));
	hipMalloc((void**)&d_Gy, 9 * sizeof(int));

	hipMemcpy(d_pos, pos, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dir, dir, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Gx, Gx, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Gy, Gy, 9 * sizeof(float), hipMemcpyHostToDevice);

	deviceComputeGradient << < BLOCKS, THREADS >> > (d_arr, d_xGradient, WIDTH, WIDTH * HEIGHT, d_Gx, d_dir, d_pos);
	deviceComputeGradient << < BLOCKS, THREADS >> > (d_arr, d_yGradient, WIDTH, WIDTH * HEIGHT, d_Gy, d_dir, d_pos);
	deviceMerge << < BLOCKS, THREADS >> > (d_xGradient, d_yGradient, d_gradient, WIDTH, WIDTH * HEIGHT);
	//deviceComputeGradient << <BLOCKS, THREADS >> >(d_arr, SIZE);

	hipMemcpy(xGradient, d_xGradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(yGradient, d_yGradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gradient, d_gradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < HEIGHT; j++) {
			/*if (i < 10 && j < 10)
				cout << gradient[i * WIDTH + j] << "\t";*/
			img_xGradient(i, j) = xGradient[i * WIDTH + j];
			img_yGradient(i, j) = yGradient[i * WIDTH + j];
			result(i, j) = gradient[i * WIDTH + j];
		}
	}
	
	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	printf("GPU for image %s takes %.2f seconds\n", filename.c_str(), duration);

	free(arr);
	free(xGradient);

	hipFree(d_arr);
	hipFree(d_xGradient);
	hipFree(d_yGradient);
	hipFree(d_gradient);

	hipFree(d_pos);
	hipFree(d_dir);
	hipFree(d_Gx);
	hipFree(d_Gy);

	(img_xGradient, img_yGradient, result).display("HOLA MUNDO CUDA");
}


int main(int argc, char** argv) {
	
	for (int i = 1; i < 2; i++)
	{
		int len = 4;
		int baseSize = 1024;
		for (int j = 1; j <= len; j++)
		{
			ostringstream stream;
			stream << (baseSize * j);
			if (i == 0)
				performCPU(stream.str() + "x" + stream.str() + ".jpg");
			else 
				performGPU(stream.str() + "x" + stream.str() + ".jpg");
		}
	}

	return 0;
}
